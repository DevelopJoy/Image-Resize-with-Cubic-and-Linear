
#include <hip/hip_runtime.h>
//双线性插值
__global__ void zoomOutIn(const int n, const float*src, int srcWidth, int srcHeight, \
	float *dst, int dstWidth, int dstHeight) {

	float srcColTidf;
	float srcRowTidf;
	float c, r;
	const float rowScale = srcHeight / (float)(dstHeight);
	const float colScale = srcWidth / (float)(dstWidth);
	//int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
		i < (n); \
		i += blockDim.x * gridDim.x) {
		int tidC = i;
		int tidR = i;// *colScaleExtend;
		float srcColTidf = (float)((tidC % (dstWidth)) * colScale);
		float srcRowTidf = (float)((tidR / (dstWidth)) * rowScale);
		int srcColTid = (int)srcColTidf;
		int srcRowTid = (int)srcRowTidf;
		c = srcColTidf - srcColTid;
		r = srcRowTidf - srcRowTid;

		int dstInd = i;
		int srcInd = srcRowTid * srcWidth + srcColTid;
		dst[dstInd] = 0;
		dst[dstInd] += (1 - c)*(1 - r)*src[srcRowTid * srcWidth + srcColTid];
		dst[dstInd] += (1 - c)*r*src[(srcRowTid + 1)*srcWidth + srcColTid];
		dst[dstInd] += c*(1 - r)*src[srcRowTid*srcWidth + srcColTid + 1];
		dst[dstInd] += c*r*src[(srcRowTid + 1)*srcWidth + srcColTid + 1];
	}
}

//双三次插值
__global__ void zoomCubicOutIn(const int n, const float*src, int srcWidth, int srcHeight, \
	float *dst, int dstWidth, int dstHeight) {

	float srcColTidf;
	float srcRowTidf;
	float c, r;
	float A = -0.75;
	const float rowScale = srcHeight / (float)(dstHeight);
	const float colScale = srcWidth / (float)(dstWidth);
	//int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
		i < (n); \
		i += blockDim.x * gridDim.x) {
		int tidC = i;
		int tidR = i;// *colScaleExtend;
		float srcColTidf = (float)((tidC % (dstWidth)) * colScale);
		float srcRowTidf = (float)((tidR / (dstWidth)) * rowScale);
		int srcColTid = (int)srcColTidf;
		int srcRowTid = (int)srcRowTidf;
		c = srcColTidf - srcColTid;
		r = srcRowTidf - srcRowTid;

		int dstInd = i;
		int srcInd = srcRowTid * srcWidth + srcColTid;
		dst[dstInd] = 0;

		{
			//
			float coeffsY[4];
			coeffsY[0] = ((A*(r + 1) - 5 * A)*(r + 1) + 8 * A)*(r + 1) - 4 * A;
			coeffsY[1] = ((A + 2)*r - (A + 3))*r*r + 1;
			coeffsY[2] = ((A + 2)*(1 - r) - (A + 3))*(1 - r)*(1 - r) + 1;
			coeffsY[3] = 1.f - coeffsY[0] - coeffsY[1] - coeffsY[2];

			float coeffsX[4];
			coeffsX[0] = ((A*(c + 1) - 5 * A)*(c + 1) + 8 * A)*(c + 1) - 4 * A;
			coeffsX[1] = ((A + 2)*c - (A + 3))*c*c + 1;
			coeffsX[2] = ((A + 2)*(1 - c) - (A + 3))*(1 - c)*(1 - c) + 1;
			coeffsX[3] = 1.f - coeffsX[0] - coeffsX[1] - coeffsX[2];

			dst[dstInd] =
				src[(srcRowTid - 1) * srcWidth + (srcColTid - 1)] * coeffsX[0] * coeffsY[0] \
				+ src[(srcRowTid)* srcWidth + (srcColTid - 1)] * coeffsX[0] * coeffsY[1] \
				+ src[(srcRowTid + 1) * srcWidth + (srcColTid - 1)] * coeffsX[0] * coeffsY[2] \
				+ src[(srcRowTid + 2) * srcWidth + (srcColTid - 1)] * coeffsX[0] * coeffsY[3] \
				+ src[(srcRowTid - 1) * srcWidth + (srcColTid)] * coeffsX[1] * coeffsY[0] \
				+ src[(srcRowTid)* srcWidth + (srcColTid)] * coeffsX[1] * coeffsY[1] \
				+ src[(srcRowTid + 1) * srcWidth + (srcColTid)] * coeffsX[1] * coeffsY[2] \
				+ src[(srcRowTid + 2) * srcWidth + (srcColTid)] * coeffsX[1] * coeffsY[3] \
				+ src[(srcRowTid - 1) * srcWidth + (srcColTid + 1)] * coeffsX[2] * coeffsY[0] \
				+ src[(srcRowTid)* srcWidth + (srcColTid + 1)] * coeffsX[2] * coeffsY[1] \
				+ src[(srcRowTid + 1) * srcWidth + (srcColTid + 1)] * coeffsX[2] * coeffsY[2] \
				+ src[(srcRowTid + 2) * srcWidth + (srcColTid + 1)] * coeffsX[2] * coeffsY[3] \
				+ src[(srcRowTid - 1) * srcWidth + (srcColTid + 2)] * coeffsX[3] * coeffsY[0] \
				+ src[(srcRowTid)* srcWidth + (srcColTid + 2)] * coeffsX[3] * coeffsY[1] \
				+ src[(srcRowTid + 1) * srcWidth + (srcColTid + 2)] * coeffsX[3] * coeffsY[2] \
				+ src[(srcRowTid + 2) * srcWidth + (srcColTid + 2)] * coeffsX[3] * coeffsY[3];
		}
	}
}
